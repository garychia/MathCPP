#include "CUDAUtilities.cuh"

#include <iostream>

void CheckCUDAStatus(const hipError_t &error)
{
    if (error != hipSuccess)
    {
        std::printf("Error: %s:%d, ", __FILE__, __LINE__);
        std::printf("code:%d, reason: %sn", error, hipGetErrorString(error));
        std::exit(1);
    }
}